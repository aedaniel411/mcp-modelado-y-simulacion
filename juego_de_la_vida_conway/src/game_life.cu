#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <string.h>
#include <time.h>
#include <ncurses.h>

#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

//#define DELAY 600000
#define DELAY 1

__global__ void inicializar (int *mat, int rows, int cols, unsigned int seed) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ren = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ren * cols + col;

	hiprandState_t state;
	hiprand_init (seed, index, 0, &state);

	mat[index] = hiprand (&state) % 2;
}

void mostrar (int *mat, int rows, int cols) {
	char cBlock = (char)0x2588;

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++ ) {
			if (mat[i * cols + j] == 1) {
				mvaddch(i, j, cBlock);
			}
		}
	}
}

__device__ int sumvivos (int *mat, int row, int rows, int col, int cols) {
	int sum = 0;

	for (int r = (row - 1); r <= row + 1; r++) {
		for (int c = col - 1; c <= col + 1; c++) {
			if (!((r == row) && (c == col))) {
				if ((r >= 0) && (r < rows)) {
					if ((c >= 0) && (c < cols)) {
						sum = sum + mat[r * cols + c];
					}
				}
			}
		}
	}

	return sum;
}

__device__ int viveomuere (int actual, int vivos) {
	int siguiente = 0;
	if (actual == 0) {
		if (vivos == 3) {				//Nacimiento
			siguiente = 1;
		}
	} else {
		siguiente = 1;
		if (vivos < 2) {
			siguiente = 0; 			//muerte por soledad
		}
		if (vivos > 3) {
			siguiente = 0;				//muerte por superpoblacion
		}
	}

	return siguiente;
}

__global__ void nuevaGeneracon (int *matOLD, int *matNEW, int rows, int cols) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int ren = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ren * cols + col;
	int sum = 0;

	sum = sumvivos (matOLD, ren, rows, col, cols);
	matNEW[index] = viveomuere(matOLD[index], sum);
}

int cuantosBloques(int N, int hilos) {
	int c, m = N % hilos;
	if (m > 0) {
		c = (N + hilos) / hilos;
	} else {
		c = N / hilos;
	}
	return c;
}

int main() {
	int rows, cols; //, sum = 0;

	initscr();
	curs_set(FALSE);

	getmaxyx(stdscr, rows, cols);
	rows = rows - 1;

	int *matA;
	matA = (int *)calloc( cols * rows, sizeof(int));

	int *dmatA, *dmatB;
	hipMalloc((void**)&dmatA, cols * rows * sizeof(int));

	dim3 dimThreadsBloque(16, 16);

	dim3 dimBloques(cuantosBloques(cols,16), cuantosBloques(rows,16));

	inicializar <<<dimBloques, dimThreadsBloque>>> (dmatA, rows, cols, time(NULL));

	hipMemcpy(matA, dmatA, cols * rows * sizeof(int), hipMemcpyDeviceToHost);

	mostrar (matA, rows, cols);

	refresh();
	getch();

	int g = 1;

	while (1) {
		hipMalloc((void**)&dmatB, cols * rows * sizeof(int));

		nuevaGeneracon <<<dimBloques, dimThreadsBloque>>> (dmatA, dmatB, rows, cols);

		hipMemcpy(matA, dmatB, cols * rows * sizeof(int), hipMemcpyDeviceToHost);

		dmatA = dmatB;
        hipFree (dmatB);
		clear();
		mostrar (matA, rows, cols);
		mvprintw (rows, 1, "Generacion %d", g++);
		refresh();

//		usleep(DELAY);
		sleep(DELAY);

	}

	free (matA);
	hipFree(dmatA);

	endwin();
	return 0;
}
